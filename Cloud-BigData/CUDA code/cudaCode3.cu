
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void add(int *a, int *b, int *c) {
  c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
  printf("The value of c is: %d \n", c[blockIdx.x]);
}

void random_ints(int* a, int N){
   int i;
   for (i = 0; i < N; ++i)
    a[i] = rand()%10;
}


int main(void) {
  int *a, *b, *c;      // host copies of a, b, c
  int *d_a, *d_b, *d_c;    // device copies of a, b, c
  int size = sizeof(int);

  a = (int*)malloc(512*size);
  b = (int*)malloc(512*size);
  c = (int*)malloc(512*size);
  
  random_ints(a,512);
  random_ints(b,512);

  // Allocate space for device copies of a, b, c
  hipMalloc((void **)&d_a, 512*size);
  hipMalloc((void **)&d_b, 512*size);
  hipMalloc((void **)&d_c, 512*size);

  // Copy inputs to device
  hipMemcpy(d_a, a, 512*size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, 512*size, hipMemcpyHostToDevice);

  // Launch add() kernel on GPU
  add<<<512,1>>>(d_a, d_b, d_c);

  // Copy result back to host
  hipMemcpy(c, d_c, 512*size, hipMemcpyDeviceToHost);

  // Cleanup
  hipFree(d_a); hipFree(d_b); hipFree(d_c);

  int i=0;
  printf("Array a: \n");
  for(i=0; i<512; i++){
    printf("%d ", a[i]);
  }
  printf("\n\n");

  printf("Array b: \n");
  for(i=0; i<512; i++){
    printf("%d ", b[i]);
  }
  printf("\n\n");

  printf("Array c: \n");
  for(i=0; i<512; i++){
    printf("%d ", c[i]);
  }
  printf("\n\n");

 
  return 0;
}
