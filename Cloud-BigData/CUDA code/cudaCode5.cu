
#include <hip/hip_runtime.h>
#include<stdio.h>


__global__ void add(int *a, int *b, int *c, int size) {
  int row=threadIdx.y;
  int col = threadIdx.x;
  c[row + (size*col)] = a[row + (size*col)] + b[row + (size*col)];
  printf("The value of c is: %d \n", c[row + (size*col)]);
}

void random_ints(int* a, int X){
   int i;
 
   for (i = 0; i < X; i++){
     
    	a[i] = rand()%10; 
     
   }
}


int main(void) {
  int *a, *b, *c;      // host copies of a, b, c
  dim3 threadM(10, 10);

  // device copies of a, b, c
  int* d_a;
  int* d_b;
  int* d_c;    
  int size = sizeof(int);

  int i=0;

  
  a = (int*)malloc(10*10*size);
  b = (int*)malloc(10*10*size);
  c = (int*)malloc(10*10*size);



  random_ints(a,10*10);
  random_ints(b,10*10);

  


  // Allocate space for device copies of a, b, c

  hipMalloc((void **)(&d_a), 10*10*size);
  hipMalloc((void **)(&d_b), 10*10*size);
  hipMalloc((void **)(&d_c), 10*10*size);
  
  // Copy inputs to device
  hipMemcpy(d_a, a, 10*10*size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, 10*10*size, hipMemcpyHostToDevice);
  
  printf("Memory coppied from host to device!\n");

  // Launch add() kernel on GPU
  add<<<1, threadM>>>(d_a, d_b, d_c, 10);

  // Copy result back to host
  hipMemcpy(c, d_c, 10*10*size, hipMemcpyDeviceToHost);
  printf("Memory coppied from device to host!\n");

  // Cleanup
  //cudaFree(*d_a); cudaFree(*d_b); cudaFree(*d_c);

  
  printf("Matrix a: \n");
  for(i=0; i<100; i++){
    if(i%10==0) printf("\n");
    printf("%d ", a[i]);
    
  }
	
  printf("\n");
  printf("Matrix b: \n");
  for(i=0; i<100; i++){
    if(i%10==0) printf("\n");
    printf("%d ", b[i]);
    
  }
  
  printf("\n");
  printf("Matrix c: \n");
  for(i=0; i<100; i++){
    if(i%10==0) printf("\n");
    printf("%d ", c[i]);
    
  }
  printf("\n\n");

 
  return 0;
}
